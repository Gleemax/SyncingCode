#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h> 

hipError_t matrixAdd(int *c, const int *a, const int *b, unsigned int sizeX, unsigned int sizeY);

__global__ void matrixAddKernel(int *g_odata, const int *g_idataA, const int *g_idataB)
{
	__shared__ int sdatai[16];
	__shared__ int sdataj[16];

    unsigned int bid = threadIdx.x;
	unsigned int tid = blockIdx.x;
	unsigned int id = bid*blockDim.x + tid;

	sdatai[id] = g_idataA[id];
	sdataj[id] = g_idataB[id];
	__syncthreads();

	g_odata[id] = sdatai[id] + sdataj[id];
}

int main()
{
    const int matrixSizeX = 4;
	const int matrixSizeY = 4;
	const int a[matrixSizeX*matrixSizeY] = 
		{ 1, 2, 3, 4, 2, 3, 4, 5, 3, 4, 5, 6, 4, 5, 6, 7 };
    const int b[matrixSizeX*matrixSizeY] = 
		{ 20, 30, 40, 50, 30, 40, 50, 60, 40, 50, 60, 70, 50, 60, 70, 80 };
    int c[matrixSizeX*matrixSizeY] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = matrixAdd(c, a, b, matrixSizeX, matrixSizeY);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixAdd failed!");
        return 1;
    }

    
	for (int i = 0; i < matrixSizeX; i++)
	{
		for (int j = 0; j < matrixSizeY; j++)
			printf("%d ",c[i*matrixSizeX+j]);
		printf("\n");
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	system("pause");

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t matrixAdd(int *c, const int *a, const int *b, unsigned int sizeX, unsigned int sizeY)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

	unsigned int num_blocks = sizeX;
	unsigned int num_threads = sizeY;
	unsigned int mem_size = sizeof(int)*sizeX*sizeY;
   
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, mem_size);
    cudaStatus = hipMalloc((void**)&dev_a, mem_size);
    cudaStatus = hipMalloc((void**)&dev_b, mem_size);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, mem_size, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, b, mem_size, hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);
	matrixAddKernel<<<grid,threads,mem_size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixAddKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, mem_size, hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
